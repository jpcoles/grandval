#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include "ic.h"

struct iclist ics[] = 
{ {ic_random,              "random",  ""},
  {NULL, NULL, NULL}
};

//==============================================================================
//                                 ic_random
//==============================================================================
void ic_random(struct particle *p, int N, pos_t R)
{
    int i;
    for (i=0; i < N; i++)
    {
        p[i].x[0] = (pos_t)(R * (2*drand48()-1));
        p[i].x[1] = (pos_t)(R * (2*drand48()-1));
        p[i].x[2] = 0; //(pos_t)(env->radius * (2*drand48()-1));
        p[i].v[0] = 0;
        p[i].v[1] = 0;
        p[i].v[2] = 0;
    }
}

//==============================================================================
//                                 ic_circular
//==============================================================================
void ic_circular(struct particle *p, int N, pos_t R)
{
    int i;
    for (i=0; i < N; i++)
    {
        p[i].x[0] = (pos_t)(R * (2*drand48()-1));
        p[i].x[1] = 0; //(pos_t)(R * (2*drand48()-1));
        p[i].x[2] = 0; //(pos_t)(env->radius * (2*drand48()-1));
        p[i].v[0] = 0;
        p[i].v[1] = sqrt(2*1e1 / fabs(p[i].x[0]));
        p[i].v[2] = 0;
    }
}
